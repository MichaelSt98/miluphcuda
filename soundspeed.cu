#include "hip/hip_runtime.h"
/**
 * @author      Christoph Schaefer cm.schaefer@gmail.com
 *
 * @section     LICENSE
 * Copyright (c) 2019 Christoph Schaefer
 *
 * This file is part of miluphcuda.
 *
 * miluphcuda is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * miluphcuda is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with miluphcuda.  If not, see <http://www.gnu.org/licenses/>.
 *
 */

#include "miluph.h"
#include "soundspeed.h"
#include "pressure.h"


__global__ void calculateSoundSpeed()
{
    register int i, inc, matId;
    int d;
    int j;
    double m_com;

    inc = blockDim.x * gridDim.x;
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i += inc) {
        matId = p_rhs.materialId[i];
        if (EOS_TYPE_POLYTROPIC_GAS == matEOS[matId]) {
            p.cs[i] = sqrt(matPolytropicK[matId] * pow(p.rho[i], matPolytropicGamma[matId]-1.0));
        } else if (EOS_TYPE_LOCALLY_ISOTHERMAL_GAS == matEOS[matId]) {
            // p = vkep \times scale_height
            double distance = 0.0;
            distance = p.x[i] * p.x[i];
#if DIM > 1
            distance += p.y[i]*p.y[i];
#if DIM > 2
            distance += p.z[i]*p.z[i];
#endif
#endif
            distance = sqrt(distance);
            m_com = 0;
            for (j = 0; j < numPointmasses; j++) {
                m_com += pointmass.m[j];
            }

            double vkep = sqrt(C_GRAVITY_SI * m_com/distance);
            p.cs[i] = vkep * scale_height;
        } else if (EOS_TYPE_IDEAL_GAS == matEOS[matId]) {
            p.cs[i] = sqrt(matPolytropicGamma[matId] * p.p[i] / p.rho[i]);
        } else if (EOS_TYPE_JUTZI == matEOS[matId] || EOS_TYPE_JUTZI_MURNAGHAN == matEOS[matId]) {
#if PALPHA_POROSITY
            //p.cs[i] = sqrt(matBulkmodulus[matId]/matTillRho0[matId]);
//            if (p.alpha_jutzi[i] > 1.0 && abs(p.dalphadp[i]) > 0) {
//                if (abs(p.delpdelrho[i]) > 0.0 || abs(p.delpdele[i]) > 0.0) {
//                    p.cs[i] = sqrt((p.alpha_jutzi[i] * p.delpdelrho[i] + p.delpdele[i] * p.p[i] / (p.rho[i] * p.rho[i]))
//                                / (p.alpha_jutzi[i] + p.dalphadp[i] * (p.p[i] - p.rho[i] * p.delpdelrho[i])));
//                }
//            }
//            if (!isnan(p.cs[i])) {
//                p_rhs.cs_old[i] = p.cs[i];
//            } else {
//                p.cs[i] = p_rhs.cs_old[i];
//            }
//#if 0
			/* switched from jutzis implementation of the soundspeed to a linear soundspeed from cs_porous with alpha=alpha0 to cs_solid with alpha=1 (also done in iSale) */
			p.cs[i] = matcs_solid[matId] + (matcs_porous[matId] - matcs_solid[matId]) * (p.alpha_jutzi[i] - 1.0) / (matporjutzi_alpha_0[matId] - 1.0);
#if DEBUG
            if (isnan(p.cs[i])) {
                printf("i %d alpha_jutzi %e delpdelrho %e delpdele %e dalphadp %e p %e rho %e\n", i, p.alpha_jutzi[i], p.delpdelrho[i], p.delpdele[i], p.dalphadp[i], p.p[i], p.rho[i]);
                        assert(0);
              }
//#endif
#endif

#endif
#if SIRONO_POROSITY
        } else if (EOS_TYPE_SIRONO == matEOS[matId]) {
            if (p.flag_plastic[i] > 0)
                p.cs[i] = sqrt(p.compressive_strength[i] / p.rho[i]);
            else
                p.cs[i] = sqrt(p.K[i] / p.rho_0prime[i]);
#endif
#if EPSALPHA_POROSITY
        /* Improvements to epsilon-alpha model by Collins et al 2010 */
        } else if (EOS_TYPE_EPSILON == matEOS[matId]) {
            double c_s0 = sqrt(matBulkmodulus[matId]/matTillRho0[matId]);
            double c_p0 = sqrt(matBulkmodulus[matId]/(matTillRho0[matId] / matporepsilon_alpha_0[matId]));
            p.cs[i] = c_s0 + (p.alpha_epspor[i] - 1.0) / (matporepsilon_alpha_0[matId] - 1.0) * (c_p0 - c_s0);
#endif
        }
        // other material types have a constant soundspeed which is calculated in initializeSoundspeed
    }
}



__global__ void initializeSoundspeed()
{
    register int i, inc, matId;
    inc = blockDim.x * gridDim.x;
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i += inc) {
        matId = p_rhs.materialId[i];
        if (EOS_TYPE_POLYTROPIC_GAS == matEOS[matId]) {
            p.cs[i] = 0.0; // for gas this will be calculated each step by kernel calculateSoundSpeed
        } else if (EOS_TYPE_ISOTHERMAL_GAS == matEOS[matId]) {
            /* this is pure molecular hydrogen at 10 K */
            p.cs[i] = 203;
        } else if (EOS_TYPE_TILLOTSON == matEOS[matId]) {
            p.cs[i] = sqrt(matBulkmodulus[matId]/matTillRho0[matId]);
        } else if (EOS_TYPE_ANEOS == matEOS[matId]) {
            p.cs[i] = aneos_bulk_cs_c[matId];
        } else if (EOS_TYPE_MURNAGHAN == matEOS[matId]) {
            p.cs[i] = sqrt(matBulkmodulus[matId]/matRho0[matId]);
        } else if (EOS_TYPE_REGOLITH == matEOS[matId]) {
            //sound speed in soil is typically between 450 and 600 m/s according to Ha H. Bui 2008
            p.cs[i] = 500.0;
//        } else if (EOS_TYPE_EPSILON == matEOS[matId]) {
//            p.cs[i] = sqrt(matBulkmodulus[matId]/matTillRho0[matId]);
        }
    }
}
