#include "hip/hip_runtime.h"
/**
 * @author      Christoph Schaefer cm.schaefer@gmail.com
 *
 * @section     LICENSE
 * Copyright (c) 2019 Christoph Schaefer
 *
 * This file is part of miluphcuda.
 *
 * miluphcuda is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * miluphcuda is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with miluphcuda.  If not, see <http://www.gnu.org/licenses/>.
 *
 */
#include "xsph.h"
#include "miluph.h"
#include "timeintegration.h"
#include "parameter.h"
#include "pressure.h"
#include "kernel.h"


extern __device__ SPH_kernel kernel;


__global__ void calculateXSPHchanges(int *interactions)
{

    register int i, k, inc, j, numInteractions;

    double W;
    double dWdx[DIM];
    double dWdr;
    double dx[DIM];
    double hbar;
    double densityi, densityj;

    double vx;
#if DIM > 1
    double vy;
#endif
#if DIM > 2
    double vz;
#endif

    double dvx;
#if DIM > 1
    double dvy;
#endif
#if DIM > 2
    double dvz;
#endif

    inc = blockDim.x * gridDim.x;
    // particle loop to smooth velocity field
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i += inc) {
        if (EOS_TYPE_IGNORE == matEOS[p_rhs.materialId[i]] || EOS_TYPE_IGNORE == p_rhs.materialId[i]) {
                continue;
        }
        numInteractions = p.noi[i];
        hbar = p.h[i];
        vx = p.vx[i];
#if DIM > 1
        vy = p.vy[i];
#if DIM == 3
        vz = p.vz[i];
#endif
#endif
        densityi = p.rho[i];
        p.xsphvx[i] = 0.0;
#if DIM > 1
        p.xsphvy[i] = 0.0;
#if DIM == 3
        p.xsphvz[i] = 0.0;
#endif
#endif
        // neighbours loop
        for (k = 0; k < numInteractions; k++) {
            j = interactions[i * MAX_NUM_INTERACTIONS + k];

            // if j is brush, continue
            if (EOS_TYPE_IGNORE == matEOS[p_rhs.materialId[j]] || EOS_TYPE_IGNORE == p_rhs.materialId[j]) {
                continue;
            }

#if VARIABLE_SML
            hbar = 0.5*(p.h[i] + p.h[j]);
#endif

            densityj = p.rho[j];
            dx[0] = p.x[i] - p.x[j];
#if DIM > 1
            dx[1] = p.y[i] - p.y[j];
#if DIM > 2
            dx[2] = p.z[i] - p.z[j];
#endif
#endif
            kernel(&W, dWdx, &dWdr, dx, hbar);

            dvx = vx - p.vx[j];
#if DIM > 1
            dvy = vy - p.vy[j];
#if DIM == 3
            dvz = vz - p.vz[j];
#endif
#endif
            p.xsphvx[i] -= p.m[j] / (0.5 * (densityi + densityj)) * W * dvx;
#if DIM > 1
            p.xsphvy[i] -= p.m[j] / (0.5 * (densityi + densityj)) * W * dvy;
#if DIM == 3
            p.xsphvz[i] -= p.m[j] / (0.5 * (densityi + densityj)) * W * dvz;
#endif
#endif
        } /* neighbours loop end */
    }  /* first particle loop end */

}
