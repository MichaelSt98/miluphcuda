#include "hip/hip_runtime.h"
/**
 * @author      Christoph Schaefer cm.schaefer@gmail.com and Thomas I. Maindl
 *
 * @section     LICENSE
 * Copyright (c) 2019 Christoph Schaefer
 *
 * This file is part of miluphcuda.
 *
 * miluphcuda is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * miluphcuda is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with miluphcuda.  If not, see <http://www.gnu.org/licenses/>.
 *
 */

#include "gravity.h"
#include "tree.h"
#include "timeintegration.h"
#include "parameter.h"
#include "miluph.h"
#include "pressure.h"

extern __device__ volatile double radius;


// add acceleration due to gravity to particle acceleration
__global__ void addoldselfgravity() 
{
	int i;

	for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i += blockDim.x * gridDim.x) {
        p.ax[i] += p.g_ax[i];
#if DIM > 1
        p.ay[i] += p.g_ay[i];
#if DIM > 2
        p.az[i] += p.g_az[i];
#endif
#endif
    }
}


// adds the acceleration due to the point masses 
__global__ void gravitation_from_point_masses()
{
    int i, inc;
    int j;
    int d;
    double r;
    double rrr;
    double dr[DIM];
    inc = blockDim.x * gridDim.x;
    // loop for point masses
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numPointmasses; i += inc) {
        for (j = 0; j < numPointmasses; j++) {
            if (i == j) continue;
            r = 0.0;
            dr[0] = pointmass.x[j] - pointmass.x[i];
#if DIM > 1
            dr[1] = pointmass.y[j] - pointmass.y[i];
#if DIM > 2
            dr[2] = pointmass.z[j] - pointmass.z[i];
#endif
#endif
            for (d = 0; d < DIM; d++) {
                r += dr[d]*dr[d];
            }
            r = sqrt(r);
            rrr = r*r*r;
            pointmass.ax[i] += C_GRAVITY_SI * pointmass.m[j] * dr[0]/(rrr);
#if DIM > 1
            pointmass.ay[i] += C_GRAVITY_SI * pointmass.m[j] * dr[1]/(rrr);
#if DIM > 2
            pointmass.az[i] += C_GRAVITY_SI * pointmass.m[j] * dr[2]/(rrr);
#endif
#endif
        }
    }


    // loop over all particles
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numRealParticles; i += inc) {
        if (p_rhs.materialId[i] == EOS_TYPE_IGNORE || matEOS[p_rhs.materialId[i]] == EOS_TYPE_IGNORE) {
            continue;
        }

        for (j = 0; j < numPointmasses; j++) {
            r = 0.0;
            dr[0] = pointmass.x[j] - p.x[i];
#if DIM > 1
            dr[1] = pointmass.y[j] - p.y[i];
#if DIM > 2
            dr[2] = pointmass.z[j] - p.z[i];
#endif
#endif
            for (d = 0; d < DIM; d++) {
                r += dr[d]*dr[d];
            }
            r = sqrt(r);
            rrr = r*r*r;
            if (r < pointmass.rmax[j] && r > pointmass.rmin[j]) {
                p.ax[i] += C_GRAVITY_SI * pointmass.m[j] * dr[0]/(rrr);
#if DIM > 1
                p.ay[i] += C_GRAVITY_SI * pointmass.m[j] * dr[1]/(rrr);
#if DIM > 2
                p.az[i] += C_GRAVITY_SI * pointmass.m[j] * dr[2]/(rrr);
#endif
#endif
            } else {
                p_rhs.materialId[i] = EOS_TYPE_IGNORE;
            }
        }
    }
}

// compute self gravity using N**2 algorithm
__global__ void direct_selfgravity() 
{
    int i, inc;
    int j;
    int d;
    double a_grav[DIM];
    double dist;
    double f;
    double dx[DIM];
    double sml;

    inc = blockDim.x * gridDim.x;
    // loop over all particles
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numRealParticles; i += inc) {
        for (d = 0; d < DIM; d++) {
            a_grav[d] = 0.0;
        }
        sml = p.h[i];
        if (p_rhs.materialId[i] == EOS_TYPE_IGNORE || matEOS[p_rhs.materialId[i]] == EOS_TYPE_IGNORE) {
            continue;
        }

        // loop over all other particles
        for (j = 0; j < numRealParticles; j++) {
            if (i == j)
                continue;

            dist = 0.0;
            dx[0] = p.x[i] - p.x[j];
#if DIM > 1
            dx[1] = p.y[i] - p.y[j];
#if DIM > 2
            dx[2] = p.z[i] - p.z[j];
#endif
#endif
            for (d = 0; d < DIM; d++) {
                dist += dx[d]*dx[d];
            }
            dist = sqrt(dist);
		    f = C_GRAVITY_SI * p.m[j]; // / (distance*distance*distance);
		    f /= dist > sml ? (dist*dist*dist) : (sml*sml*sml);
            for (d = 0; d < DIM; d++) {
                a_grav[d] -= f*dx[d];
            }
        }

		p.ax[i] += a_grav[0];
		p.g_ax[i] = a_grav[0];
#if DIM > 1
		p.ay[i] += a_grav[1];
		p.g_ay[i] = a_grav[1];
#if DIM == 3
		p.az[i] += a_grav[2];
		p.g_az[i] = a_grav[2];
#endif
#endif

    }
}

// compute self gravity using the tree
__global__ void selfgravity() 
{
	int i, child, nodeIndex, childNumber, depth;
	double px, ax, dx, f, distance;
#if DIM > 1
    double py, ay, dy;
#endif
	int currentNodeIndex[MAXDEPTH];
	int currentChildNumber[MAXDEPTH];
#if DIM == 3
	double pz, az, dz;
#endif
	double sml;
    double thetasq = theta*theta;

	__shared__ volatile double cellsize[MAXDEPTH];
	if (0 == threadIdx.x) {
		cellsize[0] = 4.0 * radius * radius;
		for (i = 1; i < MAXDEPTH; i++) {
			cellsize[i] = cellsize[i - 1] * 0.25;
		}
	}

	__syncthreads();

	for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i += blockDim.x * gridDim.x) {
		px = p.x[i];
#if DIM > 1
		py = p.y[i];
#if DIM == 3
		pz = p.z[i];
#endif
#endif
        p.g_ax[i] = 0.0;
#if DIM > 1
        p.g_ay[i] = 0.0;
#endif
        sml = p.h[i];
		ax = 0.0;
#if DIM > 1
		ay = 0.0;
#if DIM == 3
		az = 0.0;
        p.g_az[i] = 0.0;
#endif
#endif

		// start at root
		depth = 1;
		currentNodeIndex[depth] = numNodes - 1;
		currentChildNumber[depth] = 0;

		do {
			childNumber = currentChildNumber[depth];
			nodeIndex = currentNodeIndex[depth];

			while(childNumber < numChildren) {
				do {
					child = childList[childListIndex(nodeIndex, childNumber)];
					childNumber++;
				} while(child == EMPTY && childNumber < numChildren);
				if (child != EMPTY && child != i) { // dont do selfgravity with yourself!
					dx = p.x[child] - px;
					distance = dx*dx;
#if DIM > 1
					dy = p.y[child] - py;
					distance += dy*dy;
#endif
#if DIM == 3
					dz = p.z[child] - pz;
					distance += dz*dz;
#endif
					// if child is leaf or far away
					//if (child < numParticles || distance * theta > cellsize[depth]) {
					if (child < numParticles || distance * thetasq > cellsize[depth]) {
						distance = sqrt(distance);
                        //distance += 1e10;
						f = C_GRAVITY_SI * p.m[child]; // / (distance*distance*distance);
						f /= distance > sml ? (distance*distance*distance) : (sml*sml*sml);
           //             f = 0.0;
						ax += f*dx;
#if DIM > 1
						ay += f*dy;
#if DIM == 3
						az += f*dz;
#endif
#endif
					} else {
						// put child on stack
						currentChildNumber[depth] = childNumber;
						currentNodeIndex[depth] = nodeIndex;
						depth++;
                        if (depth == MAXDEPTH) {
                            printf("\n\nMAXDEPTH reached in selfgravity... this is not good.\n\n");
                            assert(depth < MAXDEPTH);
                        }
						childNumber = 0;
						nodeIndex = child;
					}
				}
			}
			depth--;
		} while(depth > 0);



		p.ax[i] += ax;
		p.g_ax[i] = ax;
#if DIM > 1
		p.ay[i] += ay;
		p.g_ay[i] = ay;
#if DIM == 3
		p.az[i] += az;
		p.g_az[i] = az;
#endif
#endif
	}
}






